#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <vector>

// CUDA headers
#include <hip/hip_runtime.h>
#include <>

using namespace std;

// CUDA kernel to convert a dense matrix to COO format
__global__ void denseToCOO(const float* denseMatrix, int numRows, int numCols, int* cooRowInd, int* cooColInd, float* cooValues, int* nnz) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;

    for (int i = tid; i < numRows * numCols; i += totalThreads) {
        int row = i / numCols;
        int col = i % numCols;
        float val = denseMatrix[i];

        if (val != 0.0) {
            /* 
            atomicAdd ensures that multiple threads won't update the same shared variable simultaneously
            in this case, it ensures nnz is syncronized
            nnz + 1
            https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#atomic-functions 
            */
            int index = atomicAdd(nnz, 1);
            cooValues[index] = val;
            cooRowInd[index] = row;
            cooColInd[index] = col;
        }
    }
}

int main() {
    // Define matrix dimensions
    int numRows = 4;
    int numCols = 3;

    // Create a dense matrix as a flat array
    // CUDA doesn't seem to support 2D arrays
    vector<float> denseMatrix = {
        1.0, 0.0, 0.0,
        0.0, 2.0, 0.0,
        0.0, 0.0, 3.5,
        0.0, 0.0, 4.0
    };

    // non-zero values and device non-zero values
    int nnz = 0;
    int* d_nnz;

    // device dense memory pointer
    float* d_denseMatrix;

    // device sparse memory pointers
    int* d_cooRowInd;
    int* d_cooColInd;
    float* d_cooValues;

    // Allocate GPU memory
    hipMalloc((void**)&d_denseMatrix, numRows * numCols * sizeof(float));
    hipMalloc((void**)&d_cooRowInd, numRows * numCols * sizeof(int));
    hipMalloc((void**)&d_cooColInd, numRows * numCols * sizeof(int));
    hipMalloc((void**)&d_cooValues, numRows * numCols * sizeof(float));
    hipMalloc((void**)&d_nnz, sizeof(int));

    // Copy data from CPU to GPU
    hipMemcpy(d_denseMatrix, denseMatrix.data(), numRows * numCols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_nnz, &nnz, sizeof(int), hipMemcpyHostToDevice);

    // Define CUDA grid and block sizes
    int blockSize = 256;
    int gridSize = (numRows * numCols + blockSize - 1) / blockSize;

    // Call the CUDA kernel to convert dense to COO format
    denseToCOO << <gridSize, blockSize >> > (d_denseMatrix, numRows, numCols, d_cooRowInd, d_cooColInd, d_cooValues, d_nnz);

    // Copy the result back from GPU to CPU
    hipMemcpy(&nnz, d_nnz, sizeof(int), hipMemcpyDeviceToHost);

    vector<int> cooRowInd(nnz);
    vector<int> cooColInd(nnz);
    vector<float> cooValues(nnz);

    hipMemcpy(cooRowInd.data(), d_cooRowInd, nnz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(cooColInd.data(), d_cooColInd, nnz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(cooValues.data(), d_cooValues, nnz * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_denseMatrix);
    hipFree(d_cooRowInd);
    hipFree(d_cooColInd);
    hipFree(d_cooValues);
    hipFree(d_nnz);

    // Print COO format
    cout << "COO Format:" << endl;
    for (int i = 0; i < nnz; i++) {
        cout << "Row: " << cooRowInd[i] << ", Col: " << cooColInd[i] << ", Value: " << cooValues[i] << endl;
    }

    return 0;
}
